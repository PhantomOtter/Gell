#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "" 
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <iostream>
#include <string>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

#include "./include/Definitions.h"
#include "./include/RandomFunctions.cuh"
#include "./include/BasicFunctions.cuh"
#include "./include/Cells.cuh"
#include "./include/Mesh.cuh"

#include "./include/Initialization.cuh"
#include "./include/FileSave.cuh"
#include "./include/PDkernel.cuh"
#include "./include/LODkernel.cuh"
#include "./include/FMkernel.cuh"

#define show_time_every_iter   false  // if false, only show every time after Save_data_gap
#define save_intermediate_Cell true
#define save_intermediate_Mesh false

// Gell version based on Thrust

int main(void)
{
	showGPUinfo(true);

	clock_t Initstart = clock();
	//std::string datapath = "D:/Gell_Data/HDS/test/";
	std::string datapath = "./data/";

	// Simu time record
	double iterduration = 0;
	double gapduration = 0;
	const int gapsavenum = Max_Simulation_Time / Save_data_gap + 1;
	const int Max_Simulation_Iter = Max_Simulation_Time / Biology_dt;

	// int cellnumrecord[Max_Simulation_Iter] = { 0 };
	int* cellnumrecord = 0;
	cellnumrecord = (int*)malloc((Max_Simulation_Iter + 1) * sizeof(int));

	// double simutimerecord[Max_Simulation_Iter] = { 0 };
	double* simutimerecord = 0;
	simutimerecord = (double*)malloc((Max_Simulation_Iter + 1) * sizeof(double));
	simutimerecord[0] = 0;

	double gapsimutimerecord[gapsavenum] = { 0 };  // simulation time without saving time                  // simulation time without saving time
	int filesaveidx = 0;        // index of how may Save_data_gap(currently 60 min) of simulation has passed 
	bool showandsave = false;   // weather Save_data_gap condition matched

	// mesh struct init
	const dim3 blockSize2d(BlockWidth2d, BlockWidth2d);
	const int gdim2d = (Voxel_num + BlockWidth2d - 1) / BlockWidth2d;
	const dim3 gridSize2d(gdim2d, gdim2d);

	const dim3 blockSize3d(BlockWidth3d, BlockWidth3d, BlockWidth3d);
	const int gdim3d = (Voxel_num + BlockWidth3d - 1) / BlockWidth3d;
	const dim3 gridSize3d(gdim3d, gdim3d, gdim3d);

	Mesh_struct O2Mesh(O2_Default_Concentration, O2_Diffusion_coef, O2_Decay_rate);
	savecsv_meshslice(&O2Mesh, datapath + "Mesh_slice_0.csv");
	// savecsv_mesh(&O2Mesh, datapath + "02_Mesh_0.csv");

	// Random num init
	hiprandState* curand_states;
	hipMalloc(&curand_states, sizeof(hiprandState) * Max_Cell_num);
	set_random_states << <(Max_Cell_num + BlockWidth1d - 1) / BlockWidth1d, BlockWidth1d >> > (curand_states);
	hipDeviceSynchronize();
	clock_t Initend = clock();
	std::cout << "Initialization Finished, takes " << (double)(Initend - Initstart) / CLOCKS_PER_SEC << std::endl;

	// Cell vector init
	srand(12345);
	int initnum = Cell_Init_num;
	int currentnum = initnum;
	cellnumrecord[0] = initnum;
	int* gcurrentnum = 0;
	hipMalloc(&gcurrentnum, sizeof(int));
	hipMemcpy(gcurrentnum, &currentnum, sizeof(int), hipMemcpyHostToDevice);

	int testcpuvalue = 1;
	hipMemcpy(&testcpuvalue, gcurrentnum, sizeof(int), hipMemcpyDeviceToHost);

	std::cout <<"Test gcurrentnum ="<< testcpuvalue <<std::endl;

	thrust::host_vector<Cell> CpuCell(Max_Cell_num);
	//Cell_Initialization(CpuCell, initnum);
	Cell_sphere_Initialization(CpuCell, initnum);
	savecsv_cell(CpuCell, currentnum, datapath + "Gell_0.csv");
	thrust::device_vector<Cell> GpuCell(Max_Cell_num);
	GpuCell = CpuCell;
	Cell* GC = thrust::raw_pointer_cast(GpuCell.data());


	MechanicsMesh_struct MecMesh;

	//Simu core
	clock_t Simustart = clock();
	//currentnum = GpuCell.size();
	std::cout << "Simulation Start -- Current Cell Num: " << currentnum << std::endl;
	std::cout << std::endl;
	clock_t showsimutimestart = clock();
	clock_t showsimutimeend = clock();
	float current_cell_time = 0.f;
	int SimuIter = 0;
	int iter_savegap = Save_data_gap / Biology_dt;


	for (float current_cell_time = 0.f; current_cell_time < Max_Simulation_Time; current_cell_time += Biology_dt) {
		SimuIter++;
		clock_t Iterstart = clock();
		//Birth Module
		currentnum = CellBirth_kernel(O2Mesh.p, GpuCell, curand_states, currentnum, gcurrentnum);

		for (float mect = 0.0; mect < Biology_dt; mect += Mechanics_dt) {

			//Force and Movement Module
			MecMesh.FM_update(GpuCell, currentnum);
			GC = thrust::raw_pointer_cast(GpuCell.data());

			for (float dift = 0.0; dift < Mechanics_dt; dift += Diffusion_dt) {
				LODsolver(O2Mesh.p, O2Mesh.E, O2Mesh.F, O2Mesh.p_rate, GC, currentnum);
				//FDMsolver(O2Mesh.p, O2Mesh.p_rate, GC, currentnum);
			}
		}

		//Death Module
		currentnum = CellDeath_kernel(GpuCell, curand_states, currentnum, gcurrentnum);

		clock_t Iterend = clock();
		iterduration = (double)(Iterend - Iterstart) / CLOCKS_PER_SEC;
		gapduration += iterduration;
		cellnumrecord[SimuIter] = currentnum;
		simutimerecord[SimuIter] = iterduration;

		//**************************** Save Show Part ****************************//

		showandsave = (SimuIter + 1) % (int)(iter_savegap) == 0;

		if (show_time_every_iter) {
			std::cout << "Simulation Iter " << SimuIter << " Finished, takes " << iterduration << "s" << std::endl;
			std::cout << "Cell Num: " << currentnum << std::endl;
			std::cout << "Accumulative Time " << (double)(Iterend - Simustart) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;
		}
		if (showandsave) {
			filesaveidx = (SimuIter + 1) / iter_savegap;
			gapsimutimerecord[filesaveidx - 1] = gapduration;

			if (save_intermediate_Cell) {
				try {
					CpuCell = GpuCell;
				}
				catch (thrust::system_error& e) {
					std::cerr << "Save Module Copy back Error: " << e.what() << std::endl;
					exit(-1);
				}
				catch (std::bad_alloc& e) {
					std::cerr << "Save Module Copy back Error: " << e.what() << std::endl;
					exit(-1);
				}
				savecsv_cell(CpuCell, currentnum, datapath + "Gell_" + std::to_string(filesaveidx) + ".csv");
			}
			if (save_intermediate_Mesh) {
				// whole mesh data
				//O2Mesh.move_to_CPU();
				// only a slice
				O2Mesh.Slice_to_CPU();
				// savecsv_mesh(&O2Mesh, datapath + "Mesh_" + std::to_string(filesaveidx) + ".csv");
				savecsv_meshslice(&O2Mesh, datapath + "Mesh_slice_" + std::to_string(filesaveidx) + ".csv");
			}
			showsimutimeend = clock();
			showGPUinfo(true);
			std::cout << "Simulation from " << filesaveidx - 1 << " to " << filesaveidx << " * " << Save_data_gap/60 << " Hour finished" << std::endl;
			std::cout << "Time " << (int)((filesaveidx - 1) * Save_data_gap / 60) << " Hour" << std::endl;
			std::cout << "Current Cell Num is " << currentnum << std::endl;
			std::cout << "( " << currentnum / 1000 << " K | " << (float)currentnum / 1000 / 1000 << " M )" << std::endl;
			std::cout << "Simulation Time consumption is " << gapduration << "s" << std::endl;
			std::cout << "Simu & Save Time consumption is " << (double)(showsimutimeend - showsimutimestart) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << "Accumulative Time " << (double)(showsimutimeend - Simustart) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;
			gapduration = 0;
			showsimutimestart = clock();
		}
		//**************************** Save Show Part ****************************//
	}


	clock_t Simuend = clock();
	std::cout << std::endl;
	std::cout << "Simulation Finished, takes " << (double)(Simuend - Simustart) / CLOCKS_PER_SEC << std::endl;


	// Save final CSV if no intermediate info is saved 
	clock_t Savestart = clock();
	if (!save_intermediate_Cell) {
		CpuCell = GpuCell;
		savecsv_cell(CpuCell, currentnum, datapath + "Gell_Simuend.csv");
	}
	if (!save_intermediate_Mesh) {
		O2Mesh.move_to_CPU();
		O2Mesh.Slice_to_CPU();
		savecsv_mesh(&O2Mesh, datapath + "Mesh_Simuend.csv");
		savecsv_meshslice(&O2Mesh, datapath + "Mesh_slice_Simuend.csv");
	}
	clock_t Saveend = clock();
	std::cout << "Final File Save Finished, takes " << (double)(Saveend - Savestart) / CLOCKS_PER_SEC << std::endl;


	//thrust::host_vector<int> cpu_CellIndex_of_StartCell_of_CellMesh = CellIndex_of_StartCell_of_CellMesh;
	//savecsv_StartCell_array(cpu_CellIndex_of_StartCell_of_CellMesh, datapath + "Gell_CellMesh_StartCell_record.csv");
	savecsv_array(cellnumrecord, simutimerecord, datapath + "Gell_Time_record.csv");
	savecsv_gaparray(cellnumrecord, gapsimutimerecord, datapath + "Gell_GapTime_record.csv");

	std::cout << "Whole Simulation and Saving process finished!" << std::endl;


	// Free Spacec
	hipFree(curand_states);
	hipFree(gcurrentnum);

	free(cellnumrecord);
	free(simutimerecord);

	system("pause");
	return 0;
}
